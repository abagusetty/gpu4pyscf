/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "gint/gint.h"
#include "gint/config.h"
#include "gint/cuda_alloc.cuh"
#include "gint/g2e.h"
#include "gint/cint2e.cuh"

#include "contract_jk.cu"
#include "gint/rys_roots.cu"
#include "gint/g2e.cu"
#include "g3c2e.cuh"
#include "g3c2e_ip2.cu"

__host__
static int GINTrun_tasks_int3c2e_ip2_jk(JKMatrix *jk, BasisProdOffsets *offsets, GINTEnvVars *envs, hipStream_t stream)
{
    int nrys_roots = envs->nrys_roots;
    int ntasks_ij = offsets->ntasks_ij;
    int ntasks_kl = offsets->ntasks_kl;
    assert(ntasks_kl < 65536*THREADSY);
    dim3 threads(THREADSX, THREADSY);
    dim3 blocks((ntasks_ij+THREADSX-1)/THREADSX, (ntasks_kl+THREADSY-1)/THREADSY);
    switch (envs->nrys_roots) {
        case 1: GINTrun_int3c2e_ip2_jk_kernel0010<<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 2: GINTint3c2e_ip2_jk_kernel<2, GSIZE2_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 3: GINTint3c2e_ip2_jk_kernel<3, GSIZE3_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 4: GINTint3c2e_ip2_jk_kernel<4, GSIZE4_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 5: GINTint3c2e_ip2_jk_kernel<5, GSIZE5_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 6: GINTint3c2e_ip2_jk_kernel<6, GSIZE6_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 7: GINTint3c2e_ip2_jk_kernel<7, GSIZE7_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 8: GINTint3c2e_ip2_jk_kernel<8, GSIZE8_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        case 9: GINTint3c2e_ip2_jk_kernel<9, GSIZE9_INT3C> <<<blocks, threads, 0, stream>>>(*envs, *jk, *offsets); break;
        default:
            fprintf(stderr, "rys roots %d\n", nrys_roots);
        return 1;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error of GINTfill_int3c2e_ip2_kernel: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}


extern "C" { __host__
int GINTbuild_int3c2e_ip2_jk(BasisProdCache *bpcache,
                 double *vj, double *vk, double *dm, double *rhoj, double *rhok,
                 int *ao_offsets, int nao, int naux, int n_dm,
                 int *bins_locs_ij, int ntasks_kl, int ncp_ij, int cp_kl_id, double omega)
{
    ContractionProdType *cp_kl = bpcache->cptype + cp_kl_id;

    int ng[4] = {0,0,1,0};

    // move bpcache to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_bpcache), bpcache, sizeof(BasisProdCache)));

    JKMatrix jk;
    jk.n_dm = n_dm;
    jk.nao = nao;
    jk.naux = naux;
    jk.dm = dm;
    jk.vj = vj;
    jk.vk = vk;
    jk.rhoj = rhoj;
    jk.rhok = rhok;
    jk.ao_offsets_i = ao_offsets[0];
    jk.ao_offsets_j = ao_offsets[1];
    jk.ao_offsets_k = ao_offsets[2];
    jk.ao_offsets_l = ao_offsets[3];

    int *bas_pairs_locs = bpcache->bas_pairs_locs;
    int *primitive_pairs_locs = bpcache->primitive_pairs_locs;

    hipStream_t streams[MAX_STREAMS];
    for (int n = 0; n < MAX_STREAMS; n++){
        checkCudaErrors(hipStreamCreate(&streams[n]));
    }

    int *idx = (int *)malloc(sizeof(int) * TOT_NF * 3);
    int *l_locs = (int *)malloc(sizeof(int) * (GPU_LMAX + 2));
    GINTinit_index1d_xyz(idx, l_locs);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_idx), idx, sizeof(int) * TOT_NF*3));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_l_locs), l_locs, sizeof(int) * (GPU_LMAX + 2)));
    free(idx);
    free(l_locs);

    for (int cp_ij_id = 0; cp_ij_id < ncp_ij; cp_ij_id++){
        int n_stream = cp_ij_id % MAX_STREAMS;

        GINTEnvVars envs;
        ContractionProdType *cp_ij = bpcache->cptype + cp_ij_id;
        GINTinit_EnvVars(&envs, cp_ij, cp_kl, ng);
        envs.omega = omega;
        if (envs.nrys_roots > 9) {
            return 2;
        }

        int ntasks_ij = bins_locs_ij[cp_ij_id+1] - bins_locs_ij[cp_ij_id];
        if (ntasks_ij <= 0) continue;

        BasisProdOffsets offsets;
        offsets.ntasks_ij = ntasks_ij;
        offsets.ntasks_kl = ntasks_kl;
        offsets.bas_ij = bas_pairs_locs[cp_ij_id];
        offsets.bas_kl = bas_pairs_locs[cp_kl_id];
        offsets.primitive_ij = primitive_pairs_locs[cp_ij_id];
        offsets.primitive_kl = primitive_pairs_locs[cp_kl_id];

        int err = GINTrun_tasks_int3c2e_ip2_jk(&jk, &offsets, &envs, streams[n_stream]);

        if (err != 0) {
            return err;
        }
    }
    for (int n = 0; n < MAX_STREAMS; n++){
        checkCudaErrors(hipStreamSynchronize(streams[n]));
        checkCudaErrors(hipStreamDestroy(streams[n]));
    }

    return 0;
}

}
