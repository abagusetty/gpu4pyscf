#include "hip/hip_runtime.h"
/*
 * Copyright 2021-2024 The PySCF Developers. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#ifdef USE_SYCL
#include "gint/sycl_device.hpp"
#else
#include <hip/hip_runtime.h>
#endif
#include "gvhf-rys/vhf.cuh"
#include "gvhf-md/md_j.cuh"

#ifdef USE_SYCL
SYCL_EXTERNAL sycl_device_global<Fold2Index[165]> s_i_in_fold2idx;
SYCL_EXTERNAL sycl_device_global<Fold3Index[495]> s_i_in_fold3idx;
#else
__constant__ Fold2Index c_i_in_fold2idx[165];
__constant__ Fold3Index c_i_in_fold3idx[495];
#endif

#ifdef USE_SYCL
SYCL_EXTERNAL __global__ void md_j_kernel(RysIntEnvVars envs, JKMatrix jk, MDBoundsInfo bounds,
                                   int threadsx, int threadsy, int tilex, int tiley, sycl::nd_item<2> &item, double *shm_mem);
SYCL_EXTERNAL __global__ void md_j_s4_kernel(RysIntEnvVars envs, JKMatrix jk, MDBoundsInfo bounds,
                                   int threadsx, int threadsy, int tilex, int tiley, sycl::nd_item<2> &item, double *shm_mem);
#else
extern __global__ void md_j_kernel(RysIntEnvVars envs, JKMatrix jk, MDBoundsInfo bounds,
                                   int threadsx, int threadsy, int tilex, int tiley);
extern __global__ void md_j_s4_kernel(RysIntEnvVars envs, JKMatrix jk, MDBoundsInfo bounds,
                                   int threadsx, int threadsy, int tilex, int tiley);
#endif

int md_j_unrolled(RysIntEnvVars *envs, JKMatrix *jk, MDBoundsInfo *bounds);

extern "C" {
int MD_build_j(double *vj, double *dm, int n_dm, int nao,
                RysIntEnvVars envs, int *scheme, int *shls_slice,
                int npairs_ij, int npairs_kl,
                int *pair_ij_mapping, int *pair_kl_mapping,
                int *pair_ij_loc, int *pair_kl_loc,
                float **qd_ij_max, float **qd_kl_max,
                float *q_cond, float cutoff,
                int *atm, int natm, int *bas, int nbas, double *env)
{
    uint16_t ish0 = shls_slice[0];
    uint16_t jsh0 = shls_slice[2];
    uint16_t ksh0 = shls_slice[4];
    uint16_t lsh0 = shls_slice[6];
    uint8_t li = bas[ANG_OF + ish0*BAS_SLOTS];
    uint8_t lj = bas[ANG_OF + jsh0*BAS_SLOTS];
    uint8_t lk = bas[ANG_OF + ksh0*BAS_SLOTS];
    uint8_t ll = bas[ANG_OF + lsh0*BAS_SLOTS];
    uint8_t order = li + lj + lk + ll;
    float *tile16_qd_ij_max = qd_ij_max[4];
    float *tile16_qd_kl_max = qd_kl_max[4];
    MDBoundsInfo bounds = {li, lj, lk, ll,
        npairs_ij, npairs_kl, pair_ij_mapping, pair_kl_mapping,
        pair_ij_loc, pair_kl_loc, tile16_qd_ij_max, tile16_qd_kl_max,
        q_cond, cutoff};

    JKMatrix jk = {vj, NULL, dm, (uint16_t)n_dm};

    if (!md_j_unrolled(&envs, &jk, &bounds)) {
        int lij = li + lj;
        int lkl = lk + ll;
        int threads_ij = scheme[0];
        int threads_kl = scheme[1];
        int gout_stride = scheme[2];
        int tilex = scheme[3];
        int tiley = scheme[4];
        switch (threads_ij) {
        case 1: bounds.qd_ij_max = qd_ij_max[0]; break;
        case 2: bounds.qd_ij_max = qd_ij_max[1]; break;
        case 4: bounds.qd_ij_max = qd_ij_max[2]; break;
        case 8: bounds.qd_ij_max = qd_ij_max[3]; break;
        case 16: bounds.qd_ij_max = qd_ij_max[4]; break;
        case 32: bounds.qd_ij_max = qd_ij_max[5]; break;
        }
        switch (threads_kl) {
        case 1: bounds.qd_kl_max = qd_kl_max[0]; break;
        case 2: bounds.qd_kl_max = qd_kl_max[1]; break;
        case 4: bounds.qd_kl_max = qd_kl_max[2]; break;
        case 8: bounds.qd_kl_max = qd_kl_max[3]; break;
        case 16: bounds.qd_kl_max = qd_kl_max[4]; break;
        case 32: bounds.qd_kl_max = qd_kl_max[5]; break;
        }
        int bsizex = threads_ij * tilex;
        int bsizey = threads_kl * tiley;
        int nsq_per_block = threads_ij * threads_kl;
        int nf3ij = (lij+1)*(lij+2)*(lij+3)/6;
        int nf3kl = (lkl+1)*(lkl+2)*(lkl+3)/6;
        int blocks_ij = (npairs_ij + bsizex - 1) / bsizex;
        int blocks_kl = (npairs_kl + bsizey - 1) / bsizey;
//        if (li == lk && lj == ll) {
//            int buflen = (order+1) * nsq_per_block
//                + threads_ij * 4 + bsizey * 4
//                + nf3ij * threads_ij + nf3kl * threads_kl
//                + (order+1)*(order+2)*(order+3)/6 * nsq_per_block;
//            buflen += max(order*(order+1)*(order+2)/6, gout_stride) * nsq_per_block;
//            md_j_s4_kernel<<<blocks, threads, buflen*sizeof(double)>>>(
//                envs, jk, bounds, threads_ij, threads_kl, tilex, tiley);
//        } else {
            int buflen = (order+1) * nsq_per_block
                + threads_ij * 4 + bsizey * 4
                + nf3ij * threads_ij * 2 + nf3kl * threads_kl * 2
                + (order+1)*(order+2)*(order+3)/6 * nsq_per_block;
            buflen += max(order*(order+1)*(order+2)/6, gout_stride) * nsq_per_block;

            #ifdef USE_SYCL
            sycl::range<2> threads(gout_stride, threads_ij*threads_kl);
            sycl::range<2> blocks(blocks_kl, blocks_ij);
            sycl_get_queue()->submit([&](sycl::handler &cgh) {
              sycl::local_accessor<double, 1> local_acc(buflen, cgh);
              cgh.parallel_for(sycl::nd_range<2>(blocks * threads, threads), [=](auto item) {
                md_j_kernel(envs, jk, bounds, threads_ij, threads_kl, tilex, tiley,
                            item, GPU4PYSCF_IMPL_SYCL_GET_MULTI_PTR(local_acc));
              });
            });
            #else
            dim3 threads(threads_ij*threads_kl, gout_stride);
            dim3 blocks(blocks_ij, blocks_kl);
            md_j_kernel<<<blocks, threads, buflen*sizeof(double)>>>(
                envs, jk, bounds, threads_ij, threads_kl, tilex, tiley);
            #endif
        }
//    }

#ifndef USE_SYCL
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in MD_build_j: %s\n", hipGetErrorString(err));
        return 1;
    }
#endif
    return 0;
}

int init_mdj_constant(int shm_size)
{
    Fold2Index i_in_fold2idx[165];
    Fold3Index i_in_fold3idx[495];
    int n2 = 0;
    int n3 = 0;
    for (int l = 0; l <= LMAX*2; ++l) {
        for (int i = 0, ijk = 0; i <= l; ++i) {
        for (int j = 0; j <= l-i; ++j, ++n2) {
            i_in_fold2idx[n2].x = i;
            i_in_fold2idx[n2].y = j;
            i_in_fold2idx[n2].fold3offset = ijk;
            for (int k = 0; k <= l-i-j; ++k, ++n3, ++ijk) {
                i_in_fold3idx[n3].x = i;
                i_in_fold3idx[n3].y = j;
                i_in_fold3idx[n3].z = k;
                i_in_fold3idx[n3].fold2yz = (l+1)*(l+2)/2 - (l-j+1)*(l-j+2)/2 + k;
            }
        } }
    }
    #ifdef USE_SYCL
    sycl_get_queue()->memcpy(s_i_in_fold2idx, i_in_fold2idx, 165*sizeof(Fold2Index)).wait();
    sycl_get_queue()->memcpy(s_i_in_fold3idx, i_in_fold3idx, 495*sizeof(Fold3Index)).wait();
    #else
    hipMemcpyToSymbol(HIP_SYMBOL(c_i_in_fold2idx), i_in_fold2idx, 165*sizeof(Fold2Index));
    hipMemcpyToSymbol(HIP_SYMBOL(c_i_in_fold3idx), i_in_fold3idx, 495*sizeof(Fold3Index));
    hipFuncSetAttribute(reinterpret_cast<const void*>(md_j_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    hipFuncSetAttribute(reinterpret_cast<const void*>(md_j_s4_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set CUDA shm size %d: %s\n", shm_size,
                hipGetErrorString(err));
        return 1;
    }
    #endif
    return 0;
}
}
