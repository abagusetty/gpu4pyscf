/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS        32

#include <cmath> // For std::sqrt

// inline double calculatenorm3d(double x, double y, double z) {
//     return std::sqrt(x * x + y * y + z * z);
// }

__global__
static void _calc_distances(double *dist, const double *x, const double *y, int m, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= m || j >= n){
        return;
    }

    double dx = x[3*i]   - y[3*j];
    double dy = x[3*i+1] - y[3*j+1];
    double dz = x[3*i+2] - y[3*j+2];
    dist[i*n+j] = std::sqrt(dx * dx + dy * dy + dz * dz);
    // dist[i*n+j] = calculatenorm3d(dx, dy, dz);
}

extern "C" {
int dist_matrix(hipStream_t stream, double *dist, const double *x, const double *y, int m, int n)
{
    int ntilex = (m + THREADS - 1) / THREADS;
    int ntiley = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntilex, ntiley);
    _calc_distances<<<blocks, threads, 0, stream>>>(dist, x, y, m, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
